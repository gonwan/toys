/* nvcc -O2 test_cuda.cu -o test_cuda */
/*
benchmark sma: size=1048576 sample=5 equal=0
sma_cpu=8ms sma_gpu=64ms
benchmark sma: size=1048576 sample=5 equal=0
sma_cpu=8ms sma_gpu=6ms
benchmark sma: size=33554432 sample=5 equal=0
sma_cpu=115ms sma_gpu=49ms
benchmark sma: size=1073741824 sample=5 equal=0
sma_cpu=1575ms sma_gpu=862ms
benchmark sma: size=1048576 sample=30 equal=0
sma_cpu=6ms sma_gpu=8ms
benchmark sma: size=33554432 sample=30 equal=0
sma_cpu=54ms sma_gpu=33ms
benchmark sma: size=1073741824 sample=30 equal=0
sma_cpu=1567ms sma_gpu=880ms
 */
#include <chrono>
#include <iostream>
#include <thread>
#include <hip/hip_runtime.h>
using namespace std;
using namespace std::chrono;
using namespace std::chrono_literals;

__global__ void sma_gpu_kernel(const int *input, int *output, int size, int sample)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        int sum = 0;
        int count = 0;
        for (int i = tid; i >= 0 && i > tid - sample; i--) {
            sum += input[i];
            count++;
        }
        output[tid] = sum / count;
    }
}

void sma_gpu(const int *input, int *output, int size, int sample)
{
    int *devInput;
    int *devOutput;
    hipMalloc((void **)&devInput, sizeof(int) * size);
    hipMalloc((void **)&devOutput, sizeof(int) * size);
    hipMemcpy(devInput, input, sizeof(int) * size, hipMemcpyHostToDevice);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    sma_gpu_kernel<<<numBlocks, blockSize>>>(devInput, devOutput, size, sample);
    hipMemcpy(output, devOutput, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipFree(devInput);
    hipFree(devOutput);
}

void sma_cpu(const int *input, int *output, int size, int sample)
{
    int sum = 0;
    int count = 0;
    for (int i = 0; i < size; i++) {
        if (count < sample) {
            sum += input[i];
            count++;
            output[i] = sum / count;
        } else {
            sum -= input[i-sample];
            sum += input[i];
            output[i] = sum / count;
        }
    }
}

void print(int *output, int size)
{
    for (int i = 0; i < size; i++) {
        cout << output[i] << " ";
    }
    cout << endl;
}

void benchmark_sma(int size, int sample) {
    int *input = (int *) malloc(sizeof(int) * size);
    for (int i = 0; i < size; i++) {
        input[i] = (int) (rand() % size);
    }
    /* begin */
    this_thread::sleep_for(200ms);
    int *output1 = (int *) malloc(sizeof(int) * size);
    memset(output1, 0, sizeof(int) * size);
    auto t0 = high_resolution_clock::now();
    sma_cpu(input, output1, size, sample);
    //print(output1, size);
    auto d1 = duration_cast<std::chrono::milliseconds>(high_resolution_clock::now() - t0);
    this_thread::sleep_for(200ms);
    int *output2 = (int *) malloc(sizeof(int) * size);
    memset(output2, 0, sizeof(int) * size);
    t0 = high_resolution_clock::now();
    sma_gpu(input, output2, size, sample);
    //print(output2, size);
    auto d2 = duration_cast<std::chrono::milliseconds>(high_resolution_clock::now() - t0);
    int b = memcmp(output1, output2, sizeof(int) * size);
    cout << "benchmark sma: size=" << size << " sample=" << sample << " equal=" << b << endl;
    cout << "sma_cpu=" << d1.count() << "ms sma_gpu=" << d2.count() << "ms" << endl;
    free(output1);
    free(output2);
}

int main()
{
    /* hot run */
    benchmark_sma(1024*1024, 5);
    /* run */
    benchmark_sma(1024*1024, 5);
    benchmark_sma(32*1024*1024, 5);
    benchmark_sma(1024*1024*1024, 5);
    benchmark_sma(1024*1024, 30);
    benchmark_sma(32*1024*1024, 30);
    benchmark_sma(1024*1024*1024, 30);
    return 0;
}
